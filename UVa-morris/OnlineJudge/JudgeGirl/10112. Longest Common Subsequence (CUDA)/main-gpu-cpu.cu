#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <assert.h>
#include <omp.h>
#include <device_functions_decls.h>
using namespace std;
#define CheckErr(status) { gpuAssert((status), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, int abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#define MAXN 60005
#define CHARSET 4
typedef unsigned short uint16;
static char A[MAXN], B[MAXN];
static int c2i[128] = {};
static char i2c[CHARSET] = {'A', 'C', 'G', 'T'};
static uint16 *cuDP;
static int *cuP;
static char *cuA, *cuB;
__device__ inline int char2int(char c) {
	switch(c) {
		case 'A': return 0;
		case 'C': return 1;
		case 'G': return 2;
		case 'T': return 3;
	}
	return 0;
}
__global__ void run(char *A, int na, char *B, int nb, int i, uint16 *dp, int *P) {
	int j = blockDim.x*blockIdx.x+threadIdx.x+1;
	if (j > nb)	return ;
	int v = char2int(A[i]); 
	int last_match = P[v*MAXN+j];
	uint16 t1 = last_match ? dp[((i&1)^1)*MAXN + last_match-1]+1 : 0;
	uint16 t2 = dp[((i&1)^1)*MAXN + j];
	dp[(i&1)*MAXN + j] = t1 > t2 ? t1 : t2;
}
int lcs_len(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
	static int P[CHARSET][MAXN];
    static uint16 dp[2][MAXN];
	A--, B--;
    #pragma omp parallel for
    for (int i = 0; i < CHARSET; i++) {
        memset(P[i], 0, sizeof(int)*(nb+1));
        for (int j = 1; j <= nb; j++)
            P[i][j] = (B[j] == i2c[i])? j : P[i][j-1];
		hipMemcpy(cuP+i*MAXN, P[i], sizeof(int)*(nb+1), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < 2; i++) {
        memset(dp[i], 0, sizeof(uint16)*(nb+1));
		hipMemcpy(cuDP+i*MAXN, dp[i], sizeof(uint16)*(nb+1), hipMemcpyHostToDevice);
	}
	hipMemcpy(cuA, A, sizeof(char)*(na+1), hipMemcpyHostToDevice);
	hipMemcpy(cuB, B, sizeof(char)*(nb+1), hipMemcpyHostToDevice);
	int bsz = 512;
	dim3 bb(bsz);
	dim3 gg((nb+bsz-1)/bsz);
	for (int i = 1; i <= na; i++)
		run<<<gg, bb>>>(cuA, na, cuB, nb, i, cuDP, cuP);
	hipMemcpy(dpf, cuDP+(na&1)*MAXN, sizeof(uint16)*(nb+1), hipMemcpyDeviceToHost);
	return dpf[nb];
}
int main() {
	for (int i = 0; i < CHARSET; i++)
		c2i[i2c[i]] = i;
	hipMalloc(&cuA, sizeof(char)*MAXN);
	hipMalloc(&cuB, sizeof(char)*MAXN);
	hipMalloc(&cuP, sizeof(int)*MAXN*4);
	hipMalloc(&cuDP, sizeof(uint16)*MAXN*2);
	static uint16 dpf[MAXN];
    while (scanf("%s %s", A, B) == 2) {
        int len = lcs_len(A, strlen(A), B, strlen(B), dpf);
        printf("%d\n", len);
    }
	hipFree(cuA);
	hipFree(cuB);
	hipFree(cuP);
	hipFree(cuDP);
    return 0;
}


