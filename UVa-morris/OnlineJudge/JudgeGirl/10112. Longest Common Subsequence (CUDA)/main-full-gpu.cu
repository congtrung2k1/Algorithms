
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#define MAXN 60005
#define CHARSET 4
typedef unsigned short uint16;
static char A[MAXN], B[MAXN];
static int c2i[128];
static char i2c[CHARSET+1] = "ACGT";
static int *cuP;
static char *cuB;
static uint16 *cuDP;
__global__ void prepare(int *P, char *B, int nb) {
	int i = threadIdx.x;
	int *p = P + i*MAXN;
	char c = "ACGT"[i];
	p[0] = 0;
	for (int j = 1; j <= nb; j++)
		p[j] = (B[j] == c) ? j : p[j-1];
}
__global__ void run(int nb, uint16 *dpIn, uint16 *dpOut, int *P) {
	int j = blockDim.x*blockIdx.x+threadIdx.x+1;
	if (j > nb)	return ;
	int pos = P[j];
	uint16 t1 = pos ? dpIn[pos-1]+1 : 0;
	uint16 t2 = dpIn[j];
	dpOut[j] = t1 > t2 ? t1 : t2;
}
int lcs_len(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
	B--;
	hipMemcpy(cuB, B, sizeof(char)*(nb+1), hipMemcpyHostToDevice);
	hipMemset(cuDP, 0, sizeof(uint16)*(nb+1));
	hipMemset(cuDP+MAXN, 0, sizeof(uint16)*(nb+1));
	
	int bsz = 1024;
	dim3 bb(bsz);
	dim3 gg((nb+bsz-1)/bsz);
	prepare<<<1, 4>>>(cuP, cuB, nb);
	for (int i = 0; i < na; i++) {
		int v = c2i[A[i]];
		run<<<gg, bb>>>(nb, cuDP+(i&1)*MAXN, cuDP+((i&1)^1)*MAXN, cuP+v*MAXN);
	}
	hipMemcpy(dpf, cuDP+(na&1)*MAXN, sizeof(uint16)*(nb+1), hipMemcpyDeviceToHost);
	return dpf[nb];
}
int main() {
	for (int i = 0; i < CHARSET; i++)
		c2i[i2c[i]] = i;
	hipMalloc(&cuB, sizeof(char)*MAXN);
	hipMalloc(&cuP, sizeof(int)*MAXN*4);
	hipMalloc(&cuDP, sizeof(uint16)*MAXN*2);
	static uint16 dpf[MAXN];
    while (scanf("%s %s", A, B) == 2) {
        int len = lcs_len(A, strlen(A), B, strlen(B), dpf);
        printf("%d\n", len);
    }
	hipFree(cuB);
	hipFree(cuP);
	hipFree(cuDP);
    return 0;
}


