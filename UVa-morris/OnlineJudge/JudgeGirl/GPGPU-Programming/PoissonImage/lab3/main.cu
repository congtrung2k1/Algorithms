#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <algorithm>
#include "SyncedMemory.h"
#include "pgm.h"
#include "lab3.h"
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

int main(int argc, char **argv)
{
	if (argc != 7) {
		printf("Usage: %s <background> <target> <mask> <offset x> <offset y> <output>\n", argv[0]);
		abort();
	}
	bool sucb, suct, sucm;
	int wb, hb, cb, wt, ht, ct, wm, hm, cm;
	auto imgb = ReadNetpbm(wb, hb, cb, sucb, argv[1]);
	auto imgt = ReadNetpbm(wt, ht, ct, suct, argv[2]);
	auto imgm = ReadNetpbm(wm, hm, cm, sucm, argv[3]);
	if (not (sucb and suct and sucm)) {
		puts("Something wrong with reading the input image files.");
		abort();
	}
	if (wt != wm or ht != hm) {
		puts("The mask and target image must have the same size.");
		abort();
	}
	if (cm != 1) {
		puts("The mask image must be mono-colored.");
		abort();
	}
	if (cb != 3 or ct != 3) {
		puts("The background and target image must be colored.");
		abort();
	}
	const int oy = atoi(argv[4]), ox = atoi(argv[5]);

	const int SIZEB = wb*hb*3;
	const int SIZET = wt*ht*3;
	const int SIZEM = wm*hm;
	MemoryBuffer<float> background(SIZEB), target(SIZET), mask(SIZEM), output(SIZEB);
	auto background_s = background.CreateSync(SIZEB);
	auto target_s = target.CreateSync(SIZET);
	auto mask_s = mask.CreateSync(SIZEM);
	auto output_s = output.CreateSync(SIZEB);

	float *background_cpu = background_s.get_cpu_wo();
	float *target_cpu = target_s.get_cpu_wo();
	float *mask_cpu = mask_s.get_cpu_wo();
	copy(imgb.get(), imgb.get()+SIZEB, background_cpu);
	copy(imgt.get(), imgt.get()+SIZET, target_cpu);
	copy(imgm.get(), imgm.get()+SIZEM, mask_cpu);

	PoissonImageCloning(
		background_s.get_gpu_ro(),
		target_s.get_gpu_ro(),
		mask_s.get_gpu_ro(),
		output_s.get_gpu_wo(),
		wb, hb, wt, ht, oy, ox
	);

	unique_ptr<uint8_t[]> o(new uint8_t[SIZEB]);
	const float *o_cpu = output_s.get_cpu_ro();
	transform(o_cpu, o_cpu+SIZEB, o.get(), [](float f) -> uint8_t { return max(min(int(f+0.5f), 255), 0); });
	WritePPM(o.get(), wb, hb, argv[6]);
	return 0;
}
