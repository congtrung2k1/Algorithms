#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
const int MAXN = 16777216;

__global__ void stage_first(int n, char *str, int32_t *pos) {
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if (idx >= n)
		return ;
	pos[idx] = str[idx] > ' ' ? -1 : idx;
}
__global__ void stage_second(int n, int32_t *pos) {
	extern __shared__ int32_t tmp[];
	int m = blockDim.x;
	int tid = threadIdx.x;
	int base = (blockDim.x*blockIdx.x)*2;
	int off = 1;
	tmp[2*tid] = pos[2*tid+base];
	tmp[2*tid+1] = pos[2*tid+1+base];
	for (int d = m; d > 0; d >>= 1, off <<= 1) {
		__syncthreads();
		if (tid < d) {
			int32_t ai = off*(2*tid+1)-1;
			int32_t bi = off*(2*tid+2)-1;
			tmp[bi] = tmp[bi] > tmp[ai] ? tmp[bi] : tmp[ai];
		}
	}
	if (tid == 0) {
		tmp[2*m] = tmp[2*m-1];
		tmp[2*m-1] = -1;
	}
	for (int d = 1; d < 2*m; d <<= 1) {
		off >>= 1;
		__syncthreads();
		if (tid < d) {
			int32_t ai = off*(2*tid+1)-1;
			int32_t bi = off*(2*tid+2)-1;
			int32_t t = tmp[ai];
			tmp[ai] = tmp[bi];
			tmp[bi] = t > tmp[bi] ? t : tmp[bi];
		}
	}
	__syncthreads();
	pos[2*tid+base] = tmp[2*tid+1];
	pos[2*tid+1+base] = tmp[2*tid+2];
}
__global__ void stage_fixed(int n, __global__ int32_t *pos, int bsz) {
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if (idx >= n || idx/bsz == 0)
		return ;
	if (pos[idx] == -1)
		pos[idx] = pos[idx/bsz*bsz-1];
}
__global__ void stage_third(int n, int32_t *pos) {
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if (idx >= n)
		return ;
	pos[idx] = idx - pos[idx];
}
void custom_sol(int n, char *cuStr, int32_t *cuPos) {
	const int bsz = 256;
	const int tsz = 512;
	dim3 bb(bsz);
	dim3 gg(((n+1)/2+bsz-1)/bsz);
	stage_first<<<(n+tsz-1)/tsz, tsz>>>(n, cuStr, cuPos);
	stage_second<<<gg, bb, (bsz*2+1)*sizeof(int32_t)>>>(n, cuPos);
	stage_fixed<<<(n+tsz-1)/tsz, tsz>>>(n, cuPos, bsz*2);
	stage_third<<<(n+tsz-1)/tsz, tsz>>>(n, cuPos);
}
template<class T> struct MM {
	char *base;
	MM(char *base): base(base) {}
	__host__ __device__ T operator()(const T& index) const { return (base[index] > ' ') ? -1 : index; };
};
template<class T> struct NN {
	int32_t *base;
	NN(int32_t *base): base(base) {}
	__host__ __device__ T operator()(const T& index) const { return index-base[index]; };
};
void thrust_sol(int n, char *cuStr, int32_t *cuPos) {
	thrust::tabulate(thrust::device, cuPos, cuPos+n, MM<int32_t>(cuStr));
	thrust::inclusive_scan(thrust::device, cuPos, cuPos+n, cuPos, thrust::maximum<int>());
	thrust::tabulate(thrust::device, cuPos, cuPos+n, NN<int32_t>(cuPos));
}
int main() {
	static char *cuStr;
	static int32_t *cuPos;
	hipMalloc(&cuStr, sizeof(char)*MAXN);
	hipMalloc(&cuPos, sizeof(int32_t)*MAXN);
	static char s[MAXN];
	static int ret[MAXN];
	int cases = 0;
	while (fgets(s, MAXN, stdin)) {
		int n = strlen(s);
		hipMemcpy(cuStr, s, sizeof(char)*n, hipMemcpyHostToDevice);
		clock_t st, ed;
		st = clock();
		for (int i = 0; i < 10; i++) {
			custom_sol(n, cuStr, cuPos);
//			thrust_sol(n, cuStr, cuPos);
		}
		hipMemcpy(ret, cuPos, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
		ed = clock() - st;
		printf ("It took me %lf seconds.\n", ((float) ed)/CLOCKS_PER_SEC/10);
		puts("Checking");
		st = clock();
		for (int i = 0, sum = 0; i < n; i++) {
//			printf("%d%c", ret[i], " \n"[i==n-1]);
			if (s[i] > ' ')
				sum++;
			else
				sum = 0;
			assert(sum == ret[i]);
		}
		ed = clock() - st;
		printf ("It took me %lf seconds.\n", ((float) ed)/CLOCKS_PER_SEC);

		printf("Case #%d: PASS\n", ++cases);
	}
	hipFree(cuPos);
	hipFree(cuStr);
	return 0;
}


