
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <omp.h>
using namespace std;
#define MAXN 60005
#define CHARSET 4
#define max(x, y) (x) > (y) ? (x) : (y)
typedef unsigned short uint16;
static char A[MAXN], B[MAXN];
static int c2i[128];
static char i2c[CHARSET+1] = "ACGT";
static int *cuP;
static char *cuB;
static uint16 *cuDP;
__global__ void prepare(int *P, char *B, int nb) {
    int i = threadIdx.x;
    int *p = P + i*MAXN;
    char c = "ACGT"[i];
    p[0] = 0;
    for (int j = 1; j <= nb; j++)
        p[j] = (B[j] == c) ? j : p[j-1];
}
__global__ void run(int nb, uint16 *dpIn, uint16 *dpOut, int *P) {
    int j = blockDim.x*blockIdx.x+threadIdx.x+1;
    if (j > nb)    return ;
    int pos = P[j];
    uint16 t1 = pos ? dpIn[pos-1]+1 : 0;
    uint16 t2 = dpIn[j];
    dpOut[j] = t1 > t2 ? t1 : t2;
}
int lcs_len_seq(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
	static uint16 dp[2][MAXN];
	memset(dp[0], 0, sizeof(uint16)*(nb+1));
	dp[1][0] = 0, dpf[0] = 0;
	for (int i = 1; i <= na; i++) {
		for (int j = 1; j <= nb; j++) {
			if (A[i-1] == B[j-1])
				dp[1][j] = dp[0][j-1]+1;
			else
				dp[1][j] = max(dp[1][j-1], dp[0][j]);
		}
		memcpy(dp[0], dp[1], sizeof(uint16)*(nb+1));
	}
	for (int i = 0; i <= nb; i++)
		dpf[i] = dp[0][i];
	return dpf[nb];
}
int lcs_len_omp(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
	static int P[CHARSET][MAXN];
	static uint16 dp[2][MAXN];
	A--, B--;
#pragma omp parallel for
	for (int i = 0; i < CHARSET; i++) {
		memset(P[i], 0, sizeof(int)*(nb+1));
		for (int j = 1; j <= nb; j++)
			P[i][j] = (B[j] == i2c[i])? j : P[i][j-1];
	}
	for (int i = 0; i < 2; i++)
		memset(dp[i], 0, sizeof(uint16)*(nb+1));
#pragma omp parallel
	for (int i = 1; i <= na; i++) {
		int *Pv = P[c2i[A[i]]];
#pragma omp for
		for (int j = 1; j <= nb; j++) {
			int last_match = Pv[j];
			uint16 tmp = last_match ? dp[i&1^1][last_match-1]+1 : 0;
			dp[i&1][j] = max(dp[i&1^1][j], tmp);
		}
	}
	for (int i = 0; i <= nb; i++)
		dpf[i] = dp[na&1][i];
	return dpf[nb];
}
int lcs_len(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
	if (max(na, nb) < 2048)
		return lcs_len_seq(A, na, B, nb, dpf);
	if (nb < 10000)
		return lcs_len_omp(A, na, B, nb, dpf);
	B--;
	hipMemcpy(cuB, B, sizeof(char)*(nb+1), hipMemcpyHostToDevice);
	hipMemset(cuDP, 0, sizeof(uint16)*(nb+1));
	hipMemset(cuDP+MAXN, 0, sizeof(uint16)*(nb+1));

	int bsz = 1024;
	dim3 bb(bsz);
	dim3 gg((nb+bsz-1)/bsz);
	prepare<<<1, 4>>>(cuP, cuB, nb);
	for (int i = 0; i < na; i++) {
		int v = c2i[A[i]];
		run<<<gg, bb>>>(nb, cuDP+(i&1)*MAXN, cuDP+((i&1)^1)*MAXN, cuP+v*MAXN);
	}
	hipMemcpy(dpf, cuDP+(na&1)*MAXN, sizeof(uint16)*(nb+1), hipMemcpyDeviceToHost);
	return dpf[nb];
}
char* alloc_str(int sz) {
	return (char *) calloc(sz, sizeof(char));
}
char* substr(const char *s, int pos, int len) {
	char *t = alloc_str(len+1);
	memcpy(t, s+pos, len);
	return t;
}
char* cat(const char *sa, const char *sb) {
	int na = strlen(sa), nb = strlen(sb);
	char *t = alloc_str(na + nb + 1);
	memcpy(t, sa, na);
	memcpy(t+na, sb, nb);
	return t;
}
char* reverse(const char *s, int len) {
	char *t = substr(s, 0, len);
	char *l = t, *r = t + len - 1;
	char tmp;
	while (l < r) {
		tmp = *l, *l = *r, *r = tmp;
		l++, r--;
	}
	return t;
}
char* find_lcs(const char *a, int na, const char *b, int nb) {
	if (na > nb) {
		const char *c; int t;
		c = a, a = b, b = c;
		t = na, na = nb, nb = t;
	}

	if (na == 0)
		return alloc_str(1);

	if (na == 1) {
		for (int i = 0; i < nb; i++) {
			if (a[0] == b[i])
				return substr(a, 0, 1);
		}
		return alloc_str(1);
	}

	static uint16 t1[MAXN];
	static uint16 t2[MAXN];
	int len = lcs_len(a, na, b, nb, t1);
	if (len == 0)
		return alloc_str(1);
	int half_len = na / 2;
	char *la = substr(a, 0, half_len);
	char *ra = substr(a, half_len, na - half_len);
	char *tb = reverse(b, nb);
	char *ta = reverse(ra, na - half_len);
	lcs_len(la, half_len, b, nb, t1);
	lcs_len(ta, na - half_len, tb, nb, t2);

	int split = -1;
	for (int i = 0; i <= nb; i++) {
		if (t1[i] + t2[nb-i] == len) {
			split = i;
			break;
		}
	}

	char *lb = substr(b, 0, split);
	char *rb = substr(b, split, nb - split);
	char *sl = find_lcs(la, half_len, lb, split);
	char *sr = find_lcs(ra, na - half_len, rb, nb - split);
	char *ret = cat(sl, sr);
	free(la), free(ra), free(ta);
	free(lb), free(rb), free(tb);
	free(sl), free(sr);
	return ret;
}
int main() {
	for (int i = 0; i < CHARSET; i++)
		c2i[i2c[i]] = i;
	hipMalloc(&cuB, sizeof(char)*MAXN);
	hipMalloc(&cuP, sizeof(int)*MAXN*4);
	hipMalloc(&cuDP, sizeof(uint16)*MAXN*2);
	static uint16 dpf[MAXN];
	while (scanf("%s %s", A, B) == 2) {
		int na = strlen(A);
		int nb = strlen(B);
		int len = lcs_len(A, na, B, nb, dpf);
		char *str = find_lcs(A, na, B, nb);
		printf("%d\n", len);
		printf("%s\n", str);
		free(str);	
	}
	hipFree(cuB);
	hipFree(cuP);
	hipFree(cuDP);
	return 0;
}


