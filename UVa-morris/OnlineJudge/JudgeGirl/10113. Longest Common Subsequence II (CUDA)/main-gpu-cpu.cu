#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <assert.h>
#include <omp.h>
#include <device_functions_decls.h>
using namespace std;
#define CheckErr(status) { gpuAssert((status), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, int abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#define MAXN 60005
#define CHARSET 4
typedef unsigned short uint16;
static char A[MAXN], B[MAXN];
static int c2i[128] = {};
static char i2c[CHARSET] = {'A', 'C', 'G', 'T'};
static uint16 *cuDP;
static int *cuP;
static char *cuA, *cuB;
__device__ inline int char2int(char c) {
	switch(c) {
		case 'A': return 0;
		case 'C': return 1;
		case 'G': return 2;
		case 'T': return 3;
	}
	return 0;
}
__global__ void run(char *A, int na, char *B, int nb, int i, uint16 *dp, int *P) {
	int j = blockDim.x*blockIdx.x+threadIdx.x+1;
	if (j > nb)	return ;
	int v = char2int(A[i]); 
	int last_match = P[v*MAXN+j];
	uint16 t1 = last_match ? dp[((i&1)^1)*MAXN + last_match-1]+1 : 0;
	uint16 t2 = dp[((i&1)^1)*MAXN + j];
	dp[(i&1)*MAXN + j] = t1 > t2 ? t1 : t2;
}
int lcs_len_seq(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
    static uint16 dp[2][MAXN];
    memset(dp[0], 0, sizeof(uint16)*(nb+1));
    dp[1][0] = 0;
#define max(x, y) (x) > (y) ? (x) : (y)
    for (int i = 1; i <= na; i++) {
        for (int j = 1; j <= nb; j++) {
            if (A[i-1] == B[j-1])
                dp[1][j] = dp[0][j-1]+1;
            else
                dp[1][j] = max(dp[1][j-1], dp[0][j]);
        }
        memcpy(dp[0], dp[1], sizeof(uint16)*(nb+1));
    }
#undef max
    for (int i = 0; i <= nb; i++)
        dpf[i] = dp[0][i];
    return dpf[nb];
}
int lcs_len(const char *A, int na, const char *B, int nb, uint16 dpf[]) {
	if (nb < 2048)
        return lcs_len_seq(A, na, B, nb, dpf);
	static int P[CHARSET][MAXN];
    static uint16 dp[2][MAXN];
	A--, B--;
    #pragma omp parallel for
    for (int i = 0; i < CHARSET; i++) {
        memset(P[i], 0, sizeof(int)*(nb+1));
        for (int j = 1; j <= nb; j++)
            P[i][j] = (B[j] == i2c[i])? j : P[i][j-1];
		hipMemcpy(cuP+i*MAXN, P[i], sizeof(int)*(nb+1), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < 2; i++) {
        memset(dp[i], 0, sizeof(uint16)*(nb+1));
		hipMemcpy(cuDP+i*MAXN, dp[i], sizeof(uint16)*(nb+1), hipMemcpyHostToDevice);
	}
	hipMemcpy(cuA, A, sizeof(char)*(na+1), hipMemcpyHostToDevice);
	hipMemcpy(cuB, B, sizeof(char)*(nb+1), hipMemcpyHostToDevice);
	int bsz = 512;
	dim3 bb(bsz);
	dim3 gg((nb+bsz-1)/bsz);
	for (int i = 1; i <= na; i++)
		run<<<gg, bb>>>(cuA, na, cuB, nb, i, cuDP, cuP);
	hipMemcpy(dpf, cuDP+(na&1)*MAXN, sizeof(uint16)*(nb+1), hipMemcpyDeviceToHost);
	return dpf[nb];
}
char* alloc_str(int sz) {
    return (char *) calloc(sz, sizeof(char));
}
char* substr(const char *s, int pos, int len) {
    char *t = alloc_str(len+1);
    memcpy(t, s+pos, len);
    return t;
}
char* cat(const char *sa, const char *sb) {
    int na = strlen(sa), nb = strlen(sb);
    char *t = alloc_str(na + nb + 1);
    memcpy(t, sa, na);
    memcpy(t+na, sb, nb);
    return t;
}
char* reverse(const char *s, int len) {
    char *t = substr(s, 0, len);
    char *l = t, *r = t + len - 1;
    char tmp;
    while (l < r) {
        tmp = *l, *l = *r, *r = tmp;
        l++, r--;
    }
    return t;
}
char* find_lcs(const char *a, int na, const char *b, int nb) {
    if (na > nb) {
        const char *c; int t;
        c = a, a = b, b = c;
        t = na, na = nb, nb = t;
    }
 
    if (na == 0)
        return alloc_str(1);
 
    if (na == 1) {
        for (int i = 0; i < nb; i++) {
            if (a[0] == b[i])
                return substr(a, 0, 1);
        }
        return alloc_str(1);
    }
 
    static uint16 t1[MAXN];
    static uint16 t2[MAXN];
    int len = lcs_len(a, na, b, nb, t1);
    if (len == 0)
        return alloc_str(1);
    int half_len = na / 2;
    char *la = substr(a, 0, half_len);
    char *ra = substr(a, half_len, na - half_len);
    char *tb = reverse(b, nb);
    char *ta = reverse(ra, na - half_len);
    lcs_len(la, half_len, b, nb, t1);
    lcs_len(ta, na - half_len, tb, nb, t2);
 
    int split = -1;
    for (int i = 0; i <= nb; i++) {
        if (t1[i] + t2[nb-i] == len) {
            split = i;
            break;
        }
    }
 
    char *lb = substr(b, 0, split);
    char *rb = substr(b, split, nb - split);
    char *sl = find_lcs(la, half_len, lb, split);
    char *sr = find_lcs(ra, na - half_len, rb, nb - split);
    char *ret = cat(sl, sr);
    free(la), free(ra), free(ta);
    free(lb), free(rb), free(tb);
    free(sl), free(sr);
    return ret;
}
int main() {
	for (int i = 0; i < CHARSET; i++)
		c2i[i2c[i]] = i;
	hipMalloc(&cuA, sizeof(char)*MAXN);
	hipMalloc(&cuB, sizeof(char)*MAXN);
	hipMalloc(&cuP, sizeof(int)*MAXN*4);
	hipMalloc(&cuDP, sizeof(uint16)*MAXN*2);
	static uint16 dpf[MAXN];
    while (scanf("%s %s", A, B) == 2) {
		int na = strlen(A);
        int nb = strlen(B);
        int len = lcs_len(A, na, B, nb, dpf);
		char *str = find_lcs(A, na, B, nb);
        printf("%d\n", len);
        printf("%s\n", str);
        free(str);	
    }
	hipFree(cuA);
	hipFree(cuB);
	hipFree(cuP);
	hipFree(cuDP);
    return 0;
}


